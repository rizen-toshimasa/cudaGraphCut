
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
struct Edge{
    Edge(){
        in_next = NULL; out_next = NULL; rev = NULL;
        push = NULL; pull = NULL;
        flow = 0;
    }
    struct Edge *in_next, *out_next, *rev;
    struct Node *push, *pull;
    unsigned int flow;
};
struct Node{
    Node(){
        in_edge = NULL; out_edge = NULL;
        flow = 0; height = 0;
    }
    struct Edge *in_edge, *out_edge;
    unsigned int flow ,height;
};
void Push(struct Node *node_push, struct Node *node_pull, int flow){
    struct Edge *edge = new struct Edge;
    edge->in_next = node_pull->in_edge;
    edge->out_next = node_push->out_edge;
    node_pull->in_edge = edge;
    node_push->out_edge = edge;
    edge->pull = node_pull;
    edge->push = node_push;
    edge->flow = flow;
}
struct Edge* FindEdge(struct Node *node_push, struct Node *node_pull){
    //エッジすでにあるかなハート
    puts("ie");
    printf("%p",node_push);
    struct Edge *edge = node_push->out_edge;
    printf("node_push->out",node_push->out_edge);
puts("hoge");
int i=0;
//    while(edge != NULL){
        printf("%d\n",i);
        if(edge->pull == node_pull){
            return edge;
        }
        edge = edge->out_next;
        i++;
//    }
    return NULL;//なかったわ
}

int main(int argc, char** argv){
    FILE *fp;
    const char fname[] = "sample.inp";
    if((fp = fopen(fname,"r")) == NULL){
        printf("ファイル読み込みエラー\n");
        exit(EXIT_FAILURE);
    }
    int size_nodes, num_edge;
    fscanf(fp, "p max %d %d", &size_nodes, &num_edge);
    printf("Node:%d Edge:%d\n", size_nodes, num_edge);
    int num_src;
    int num_sink;
    fscanf(fp, "n %d s", &num_src);
    fscanf(fp, "n %d t", &num_sink);
    struct Node *nodes;
    nodes = (struct Node *)malloc(sizeof (struct Node) * size_nodes);
    int id_push, id_pull, flow;
    struct Node *node_push, *node_pull;
    struct Edge *find_edge;
    while(fscanf(fp, "a %d %d %d",&id_push, &id_pull, &flow) != EOF){
        puts("a");
        //NodeHoge(nodes, num_max_node, push, pull, flow);
        printf("id_push:%d pull:%d",id_push,id_pull);
        node_push = &nodes[id_push-1];
        node_pull = &nodes[id_pull-1];
        puts("hoge");
        find_edge = FindEdge(node_push, node_pull);
        puts("aho");
        if(find_edge == NULL){
            find_edge->pull = node_pull;
            find_edge->push = node_push;
            find_edge->flow = flow;
        }else{
            Push(node_push, node_pull, flow);
        }
    }
    free(nodes);
    fclose(fp);
    return 0;
}
