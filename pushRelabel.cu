
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
struct data{
    unsigned int o;//overflow value
    unsigned int h;//height value
    unsigned char f;//first flg
};
struct node{
    struct node *next;
    struct node *prev;
    struct data *data;
};
struct node *top;
struct node *btm;
void enqueue(struct data *data){
    struct node *node = new struct node;
    node->data = data;
    node->prev = NULL;
    if (top == NULL) btm = node;
    else top-> prev = node;
    node->next = top;
    top = node;
}
struct data  dequeue(void){
        struct node *node = btm;
        struct data *data = node->data;
        btm = node->prev;
        if(btm ==NULL) top = NULL;
        else btm->next = NULL;
        delete node;
        return *data;
}
int main(int argc, char** argv){
    FILE *fp;
    const char fname[] = "sample.inp";
    if((fp = fopen(fname,"r")) == NULL){
        printf("ファイル読み込みエラー\n");
        exit(EXIT_FAILURE);
    }
    char str[256];
    while(fgets(str, 256, fp) != NULL){
        printf("%s",str);
    }
    fclose(fp);
    return 0;
}
