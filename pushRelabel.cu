#include "hip/hip_runtime.h"
#include <stdio'/home/toshimasa/android-studio/bin/studio.sh' .h>
#include <stdlib.h>
#include <time.h>
typedef struct Node{
    struct Edge *in_edge = NULL;
    struct Edge *out_edge = NULL;
    unsigned int flow = 0;
    unsigned int height = 0;
} node_t;
typedef struct Edge{
    struct Edge *in_next = NULL;
    struct Edge *out_next = NULL;
    struct Node *push = NULL;
    struct Node *pull = NULL;
    struct Edge *rev = NULL;
    unsigned int flow = 0;
} edge_t;
void Push(struct Node *node_push, struct Node *node_pull, int flow){
    struct Edge *edge = new struct Edge;
    edge->in_next = node_pull->in_edge;
    edge->out_next = node_push->out_edge;
    node_pull->in_edge = edge;
    node_push->out_edge = edge;
    edge->pull = node_pull;
    edge->push = node_push;
    edge->flow = flow;
}
struct Edge FindEdge(struct Node *node_push, struct Node *node_pull){
    //エッジすでにあるかなハート
    struct Edge edge = node_push->out_edge;
    while(edge != NULL){
        if(edge->pull == node_pull) return edge;
    }
    return NULL;//なかったわ
    
}

int main(int argc, char** argv){
    FILE *fp;
    const char fname[] = "sample.inp";
    if((fp = fopen(fname,"r")) == NULL){
        printf("ファイル読み込みエラー\n");
        exit(EXIT_FAILURE);
    }
    int size_nodes, num_edge;
    fscanf(fp, "p max %d %d", &size_nodes, &num_edge);
    printf("Node:%d Edge:%d\n", size_nodes, num_edge);
    int num_src;
    int num_sink;
    fscanf(fp, "n %d s", &num_src);
    fscanf(fp, "n %d t", &num_sink);
    struct Node *nodes;
    nodes = (struct Node *)malloc(sizeof (struct Node) * node_max_num);
    int push, pull, flow;
    while(fscanf(fp, "a %d %d %d",&node_push, &node_pull, &flow) == NULL){
        //NodeHoge(nodes, num_max_node, push, pull, flow);
        if(struct Edge edge = FindEdge(node_push, node_pull)){
            edge->pull = pull;
            edge->push = push;
            edge->flow = flow;
        }else{
            Push(node_push, node_pull, flow);
        }
    }
    free(edgeArray)
    fclose(fp);
    return 0;
}
