#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
typedef struct Node{
    struct Edge *in_edge;
    struct Edge *out_edge;
    unsigned int flow;
    unsigned int height;
} node_t;
typedef struct Edge{
    struct Edge *in_next;
    struct Edge *out_next;
    struct Node *push;
    struct Node *pull;
    struct Edge *rev;
    struct int flow;
} edge_t;
void Push(struct Node *node_push, struct Node *node_pull, int flow){
    struct Edge edge = new edge;
    struct Edge temp_edge = node_push->out_edge;
    edge.in_next = 1;
}
bool IsExistEdge(struct Node node_push, struct Node node_pull){
    
}

int main(int argc, char** argv){
    FILE *fp;
    const char fname[] = "sample.inp";
    if((fp = fopen(fname,"r")) == NULL){
        printf("ファイル読み込みエラー\n");
        exit(EXIT_FAILURE);
    }
    int size_nodes, num_edge;
    fscanf(fp, "p max %d %d", &size_nodes, &num_edge);
    printf("Node:%d Edge:%d\n", size_nodes, num_edge);
    int num_src;
    int num_sink;
    fscanf(fp, "n %d s", &num_src);
    fscanf(fp, "n %d t", &num_sink);
    struct Node *nodes;
    nodes = (struct Node *)malloc(sizeof (struct Node) * node_max_num);
    int push, pull, flow;
    while(fscanf(fp, "a %d %d %d",&node_push, &node_pull, &flow) == NULL){
        //NodeHoge(nodes, num_max_node, push, pull, flow);
        if(!IsExistEdge(node_push, node_pull)) continue;
        Push(node_push, node_pull, flow);
    }
    free(edgeArray)
    fclose(fp);
    return 0;
}
